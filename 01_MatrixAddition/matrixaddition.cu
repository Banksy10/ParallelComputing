// 系统头文件
#include <stdlib.h>
#include <stdio.h>

// cuda头文件
#include <hip/hip_runtime.h>


#define N 10
#define BLOCK_SIZE 8


// GPU 上的向量加法，看作一维数组相加
__global__ void vectorAdd(float *a, float *b, float *c) {
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int i = bx * BLOCK_SIZE + tx;
	c[i] = a[i] + b[i];
}


// 初始化向量为随机数值
void randomInit(float* data, unsigned int size) {
	srand(1);
	for (unsigned int i = 0; i < size; i++) {
		data[i] = rand() / (float) 10000;
	}	
}


// 主机端主函数
int main(void) {
	float *aH, *bH, *cH, *aD, *bD, *cD;
	
	int mem_size = N * N * sizeof(float);

	// 在主机内存申请 A，B，C 向量的空间
	aH = (float*) malloc(mem_size);
	bH = (float*) malloc(mem_size);
	cH = (float*) malloc(mem_size);

	// 在 GPU 设备申请 A，B，C 向量的空间
	hipMalloc((void**) &aD, mem_size);
	hipMalloc((void**) &bD, mem_size);
	hipMalloc((void**) &cD, mem_size);

	// 初始化主机内存的 A，B 向量
	randomInit(aH, N * N);
	randomInit(bH, N * N);

	// 拷贝主机内存的 A，B 的内容到 GPU 设备的 A，B
	hipMemcpy(aD, aH, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(bD, bH, mem_size, hipMemcpyHostToDevice);

	// GPU 内核函数的维度参数
	dim3 dimBlock(BLOCK_SIZE, 1);
	dim3 dimGrid((N * N + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

	// 执行 GPU 内核函数
	vectorAdd <<<dimGrid, dimBlock >>> (aD, bD, cD);

	// 从 GPU 设备复制结果向量 C 到主机内存的 C
	hipMemcpy(cH, cD, mem_size, hipMemcpyDeviceToHost);
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			printf("%.2f\t", cH[i * N + j]);
		}
		printf("\n");
	}

	free(aH);
	free(bH);
	free(cH);
	hipFree(aD);
	hipFree(bD);
	hipFree(cD);
	system("pause");
}

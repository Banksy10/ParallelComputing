#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void convolution_1D_basic_kernel(float* N, float* M, float* P, int MASK_WIDTH, int Width) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float Pvalue = 0;
    int n_start_point = i - (MASK_WIDTH / 2);

    if (i < Width) {
        for (int j = 0; j < MASK_WIDTH; j++) {
            if ((n_start_point + j) >= 0 && (n_start_point + j < Width)) {
                Pvalue += N[n_start_point + j] * M[j];
            }
        }
        P[i] = Pvalue;
    }
}


int main() {
    int nSize = 7, mSize = 5;
    float* host_N, * host_P, * device_N, * device_M, * device_P;
    float host_M[5] = { 3, 4, 5, 4, 3 };

    host_N = (float*)malloc(nSize * sizeof(float));
    host_P = (float*)malloc(nSize * sizeof(float));
    for (int i = 0; i < nSize; i++) {
        host_N[i] = i * 1.0 + 1;
    }

    hipMalloc((void**)&device_N, nSize * sizeof(float));
    hipMalloc((void**)&device_M, mSize * sizeof(float));
    hipMalloc((void**)&device_P, nSize * sizeof(float));
    hipMemcpy(device_N, host_N, nSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_M, host_M, mSize * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(nSize);
    dim3 blocksPerGrid(1);

    convolution_1D_basic_kernel <<< blocksPerGrid, threadsPerBlock >>> (device_N, device_M, device_P, mSize, nSize);
    hipDeviceSynchronize();

    hipMemcpy(host_P, device_P, nSize * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < nSize; ++i) {
        printf("%.f\t", host_P[i]);
    }
    printf("\n");

    hipFree(device_N);
    hipFree(device_M);
    hipFree(device_P);
    free(host_N);
    free(host_P);
    return 0;
}


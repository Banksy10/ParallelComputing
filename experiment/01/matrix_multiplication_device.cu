#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 1024

__global__ void matrixMultiplication(float *matrixM, float *matrixN, float *matrixP, int width) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float sum = 0, m, n;
    for (int k = 0; k < width; ++k) {
        m = matrixM[ty * width + k];
        n = matrixN[k * width + tx];
        sum += m * n;
    }
    matrixP[ty * width + tx] = sum;
}

int main(void) {
    float *h_matrixM, *h_matrixN, *h_matrixP, *d_matrixM, *d_matrixN, *d_matrixP;
    int total = N * N;
    int mem_size = total * sizeof(float);

    h_matrixM = (float *) malloc(mem_size);
    h_matrixN = (float *) malloc(mem_size);
    h_matrixP = (float *) malloc(mem_size);
    hipMalloc((void **) &d_matrixM, mem_size);
    hipMalloc((void **) &d_matrixN, mem_size);
    hipMalloc((void **) &d_matrixP, mem_size);

    for (int i = 0; i < total; ++i) {
        h_matrixM[i] = 3;
        h_matrixN[i] = 2;
    }

    hipMemcpy(d_matrixM, h_matrixM, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_matrixN, h_matrixN, mem_size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(512);
    dim3 blocksPerGrid(16);
    
    // 记录程序开始运行的时间
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    matrixMultiplication <<< blocksPerGrid, threadsPerBlock >>> (d_matrixM, d_matrixN, d_matrixP, N);

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time elapsed: %.6f ms\n", time);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_matrixP, d_matrixP, mem_size, hipMemcpyDeviceToHost);

    // 输出结果
    // for (int i = 0; i < SIZE * SIZE; ++i) {
    //     printf("%.2f\n", h_matrixP[0]);
    // }

    free(h_matrixM);
    free(h_matrixN);
    free(h_matrixP);
    hipFree(d_matrixM);
    hipFree(d_matrixN);
    hipFree(d_matrixP);
    return 0;
}

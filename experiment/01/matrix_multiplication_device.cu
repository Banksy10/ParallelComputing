#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 1024

__global__ void matrixMultiplication(float *matrixM, float *matrixN, float *matrixP) {
    int bx = blockIdx.x; 
    int tx = threadIdx.x;
    float sum = 0, m, n;
    for (int k = 0; k < N; ++k) {
        m = matrixM[bx * N + k];
        n = matrixN[k * N + tx];
        sum += m * n;
    }
    matrixP[bx * N + tx] = sum;
}

int main(void) {
    float *h_matrixM, *h_matrixN, *h_matrixP, *d_matrixM, *d_matrixN, *d_matrixP;
    int total = N * N;
    int mem_size = total * sizeof(float);

    h_matrixM = (float *) malloc(mem_size);
    h_matrixN = (float *) malloc(mem_size);
    h_matrixP = (float *) malloc(mem_size);
    hipMalloc((void **) &d_matrixM, mem_size);
    hipMalloc((void **) &d_matrixN, mem_size);
    hipMalloc((void **) &d_matrixP, mem_size);

    for (int i = 0; i < total; ++i) {
        h_matrixM[i] = 3;
        h_matrixN[i] = 2;
    }

    hipMemcpy(d_matrixM, h_matrixM, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_matrixN, h_matrixN, mem_size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(1024);
    dim3 blocksPerGrid(1024);
    
    // 记录程序开始运行的时间
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    matrixMultiplication <<< blocksPerGrid, threadsPerBlock >>> (d_matrixM, d_matrixN, d_matrixP);

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time elapsed: %.6f ms\n", time);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_matrixP, d_matrixP, mem_size, hipMemcpyDeviceToHost);

    // 输出结果
    //for (int i = 0; i < N * N; ++i) {
    //    printf("h_matrixP[%d] = %.6f\n", i, h_matrixP[i]);
    //}

    free(h_matrixM);
    free(h_matrixN);
    free(h_matrixP);
    hipFree(d_matrixM);
    hipFree(d_matrixN);
    hipFree(d_matrixP);
    return 0;
}
